#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include<stdlib.h>
#include<time.h>
#include<chrono>
#include<string.h>

#define MATRIX_SCALE 1024

int g_tile_size = 32;
int g_handle_size = 4;

__constant__ int constant_tile_size;
int src_matrix[MATRIX_SCALE][MATRIX_SCALE];
int dest_matrix[MATRIX_SCALE][MATRIX_SCALE];
int dest_matrix_cpu_transpose[MATRIX_SCALE][MATRIX_SCALE];
/// <summary>
/// 矩阵拷贝的kernel
/// </summary>
/// <param name="input_dest">目的矩阵</param>
/// <param name="input_src">源矩阵</param>
/// <returns></returns>
__global__ void cuda_copy_kernel(int* input_dest,int* input_src) {//矩阵拷贝（baseline）
    int x = blockIdx.x * constant_tile_size + threadIdx.x;
    int y = blockIdx.y * constant_tile_size + threadIdx.y;
    int matrix_width = constant_tile_size * blockDim.x;
    for (int i = 0; i < constant_tile_size; i+=blockDim.y) {
        input_dest[(y + i) * matrix_width + x]= input_src[(y + i) * matrix_width + x] ;
    }
    
}
/// <summary>
/// cuda实现矩阵的拷贝
/// </summary>
/// <param name="input_dest">目的矩阵</param>
/// <param name="input_src">源矩阵</param>
/// <param name="number">总共的矩阵元素的个数</param>
void cuda_copy_matrix(int* input_dest,int* input_src,unsigned int number){
    int* matrix_input = nullptr;
    int* matrix_output = nullptr;
    /*分配内存*/
    hipMalloc((void**)&matrix_input, sizeof(int) * number);
    hipMalloc((void**)&matrix_output, sizeof(int) * number);
    /*拷贝数据*/
    hipMemcpy(matrix_input, input_src, sizeof(int) * number, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(constant_tile_size), &g_tile_size, sizeof(int),0,hipMemcpyHostToDevice);
    /*插入cude_event计时*/
    hipEvent_t start,stop;
    float ElpausedTime=0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    /*准备使用kernel*/
    dim3 gridDim(32, 32, 1);
    dim3 blockDim(g_tile_size, g_tile_size/g_handle_size, 1);
    cuda_copy_kernel << <gridDim, blockDim>> > (matrix_output, matrix_input);
    /*计算时间*/
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ElpausedTime, start, stop);
    std::cout << "copy消耗时间为" << ElpausedTime <<"ms" << std::endl;
    /*数据同步*/
    hipDeviceSynchronize();
    /*返回数据*/
    hipMemcpy(input_dest, matrix_output, sizeof(int) * number, hipMemcpyDeviceToHost);
    hipFree(matrix_input);
    hipFree(matrix_output);

}
/// <summary>
/// 随机生成元素
/// </summary>
/// <param name="input_matrix">目的矩阵</param>
/// <param name="number">矩阵的元素个数</param>
void random_generate(int * input_matrix,unsigned int number) {
    srand(time(NULL));
    for (int i = 0; i < number; i++) {
        input_matrix[i] = rand() % 100;
    }
}
/// <summary>
/// 打印矩阵
/// </summary>
/// <param name="input_matrix">目的矩阵</param>
/// <param name="number">矩阵元素的个数</param>
void print_matrix(const int* input_matrix, unsigned int number) {
    srand(time(NULL));
    for (int i = 0; i < number; i++) {
        std::cout << input_matrix[i]<<std::endl;
    }
}
/// <summary>
/// cpu版本的矩阵转置，并计时
/// </summary>
/// <param name="input_dest_matrix">目标矩阵</param>
/// <param name="input_src_matrix">源矩阵</param>
/// <param name="rows">行数</param>
/// <param name="columns">列数</param>
void transpose_matrix_cpu(int* input_dest_matrix, const int* input_src_matrix, unsigned int rows,unsigned int columns) {
    auto start_time = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < rows; i++) {
        for (int k = 0; k < columns; k++) {
            input_dest_matrix[i*rows+k] = input_src_matrix[k*columns+i];
        }
    }
    auto end_time = std::chrono::high_resolution_clock::now();
    double elapsed_time = std::chrono::duration<double, std::milli>(end_time - start_time).count();
    std::cout << "cpu transpose consume:" << elapsed_time << "ms" << std::endl;
}
/// <summary>
/// 检查矩阵是不是一样
/// </summary>
/// <param name="input_dest_matrix">目标矩阵</param>
/// <param name="input_src_matrix">源矩阵</param>
/// <param name="number">总个数</param>
void check_matrix(const int* input_dest_matrix, const int* input_src_matrix,const unsigned int number) {
    //比较
    if (std::memcmp(input_dest_matrix, input_src_matrix, number) == 0) {
        std::cout << "matrix check sucessful" << std::endl;
        return;
    }
    //纠错
    for (int i = 0; i < number; i++) {
        if (input_dest_matrix[i] != input_src_matrix[i]) {
            std::cout << "erro in " << i << std::endl;
            return;
        }
    }
}

int main()
{
    random_generate(src_matrix[0], MATRIX_SCALE * MATRIX_SCALE);
    cuda_copy_matrix(dest_matrix[0], src_matrix[0], MATRIX_SCALE * MATRIX_SCALE);//cuda矩阵拷贝
    check_matrix(dest_matrix[0], src_matrix[0], MATRIX_SCALE * MATRIX_SCALE);
    transpose_matrix_cpu(dest_matrix_cpu_transpose[0], src_matrix[0], MATRIX_SCALE, MATRIX_SCALE);//cpu版本transpose

    return 0;
}


